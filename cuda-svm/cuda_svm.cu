#include "hip/hip_runtime.h"
#include "cuda_svm.h"
// #include <hip/hip_runtime.h>
#include <time.h>

const double EPS = 0.0001;

//possible optimization: load x[][i] into shared and parallel on j
// x[coords][objs], y[objs], out[objs][objs]
__global__ static
void calc_linear_kernel(int objs,int coords,double* x,double* out){
    int id=blockDim.x * blockIdx.x + threadIdx.x;
    int i=id/objs;
    int j=id%objs;
    if (i<objs){
        
        double r=0.0;
        for (int k=0;k<coords;k++){
            r+=x[objs*k+i]*x[objs*k+j];
        }
        out[objs*i+j]=r;
    }
}

__global__ static
void calc_e(int objs,double* a,double b,int* y,double* kval,double* e){
    int id=blockDim.x * blockIdx.x + threadIdx.x;
    if (id<objs){
        double fx=b;
        for (int i=0;i<objs;i++){
            //access to a and y are not coalesced
            fx+=a[i]*y[i]*kval[i*objs+id];
        }
        e[id]=fx-y[id];
    }
}

// solid 13 arguments......
__global__ static
void update_e(int objs,double* e,double* kval,double b_old,double b_new,int i,int j,int yi,int yj,double ai_old,double ai_new,double aj_old,double aj_new){
    int id=blockDim.x * blockIdx.x + threadIdx.x;
    if (id<objs){
        double val=e[id];
        val+=(b_new-b_old);
        double ti=yi*kval[i*objs+id];
        double tj=yj*kval[j*objs+id];
        val += ti*(ai_new-ai_old);
        val += tj*(aj_new-aj_old);
        e[id]=val;
    }
}

void calLH(double C,int yi, int yj, double ai, double aj, double &lb, double &rb)  {
	if (yi != yj) {
		lb = max(0., aj - ai);
		rb = min(C, C + aj - ai);
	}
	else {
		lb = max(0., ai + aj - C);
		rb = min(C, ai + aj);
	}
}

double computeAjAndClipValue(double* a, int* y,double ei, double ej, double ita, double lb, double rb, int j) {
	double aj = a[j] - y[j] * (ei - ej) / ita;
	if (aj > rb) aj = rb;
	else if (aj < lb) aj = lb;
	return aj;
}

double determineAi(double* a,int* y,int i, int j, double aj_old) {
	return a[i] + y[i] * y[j] * (aj_old - a[j]);
}

void compute_eVal(int objs,double b, double* a_d,int* y_d,double* kval_d,double* eVal_d,double* eVal){
    //Might be costly to copy all of a since only a tiny part of a is updated
    calc_e<<<objs/256+1,256>>>(objs,a_d,b,y_d,kval_d,eVal_d);
    hipMemcpy(eVal,eVal_d,sizeof(double)*objs,hipMemcpyDeviceToHost);
}

inline void update_eVal(int objs,double* eVal_d,double* kval_d,double b_old,double b_new,int i,int j,int yi,int yj,double ai_old,double ai_new,double aj_old,double aj_new,double* eVal){
    //Might be costly to copy all of a since only a tiny part of a is updated
    update_e<<<objs/256+1,256>>>(objs,eVal_d,kval_d,b_old,b_new,i,j,yi,yj,ai_old,ai_new,aj_old,aj_new);
    hipMemcpy(eVal,eVal_d,sizeof(double)*objs,hipMemcpyDeviceToHost);
}

// loss = regularization(a) + C * loss(a; x,y)
// C ranges from 1e-5 ~ 1e5
void cuda_svm(int objs,int coords,double** x,int* y,double c,int max_passes,double* a,double* b_out){
    int b=0;

    int pass=0;

    double init_clk=(double)clock()/CLOCKS_PER_SEC;
    double** kval;
    malloc2D(kval,objs,objs,double);
    double* kval_d;
    hipMalloc(&kval_d, objs*objs*sizeof(double));
    double* a_d;
    hipMalloc(&a_d,objs*sizeof(double));
    hipMemcpy(a_d,a,objs*sizeof(double),hipMemcpyHostToDevice);

    //May possibly use Constant memory, if switched to byte*
    int* y_d;
    hipMalloc(&y_d,objs*sizeof(int));
    hipMemcpy(y_d,y,objs*sizeof(int),hipMemcpyHostToDevice);

    //Pre calculate kernel via cuda
    {
        double** x_r;
        malloc2D(x_r,coords,objs,double);
        for (int i=0;i<coords;i++)
            for (int j=0;j<objs;j++)
                x_r[i][j]=x[j][i];
        
        double* x_r_d;
        hipMalloc(&x_r_d, coords*objs*sizeof(double));
        hipMemcpy(x_r_d,x_r[0],coords*objs*sizeof(double),hipMemcpyHostToDevice);

        calc_linear_kernel<<<objs*objs/256+1,256>>>(objs,coords,x_r_d,kval_d);
        hipMemcpy(kval[0],kval_d,objs*objs*sizeof(double),hipMemcpyDeviceToHost);

        free(x_r[0]);
        free(x_r);
        hipFree(x_r_d);
    }
    printf("Training.calcKrnel time=%lf s\n",(double)clock()/CLOCKS_PER_SEC-init_clk);

    // FILE* fk=fopen("custom.txt","w");
    // for (int i=0;i<objs;i++){
    //     double ei = calE(objs,a,b,y,kval,i);
    //     fprintf(fk,"e[%d]=%f\n",i,ei);
    // }
    // fclose(fk);

    // for (int i=0;i<objs;++i)
    //   for (int j=0;j<objs;++j)
    //     printf("%.2f ",kval[i][j]);

    double* eVal=(double*)calloc(objs,sizeof(double));
    double* eVal_d;
    hipMalloc(&eVal_d,objs*sizeof(double));

    compute_eVal(objs,b,a_d,y_d,kval_d,eVal_d,eVal);


    int iter=0;
    const int max_iter=50;
    while (pass < max_passes && iter < max_iter) {
        double st_clk=(double)clock()/CLOCKS_PER_SEC;
		int num_changed_alphas = 0;
		for (int i=0; i<objs; i++) {
            double ei = eVal[i];
            // printf("e[%d]=%f\n",i,ei);
			if ((y[i]*ei < -EPS && a[i] < c) || (y[i]*ei > EPS && a[i] > 0)) {
                //updated rand method
                int j = rand() % (objs-1);
                j=(j>=i)?j+1:j;

				double ej = eVal[j];
				double ai_old = a[i], aj_old = a[j];
				double lb = 0, rb = c;
				calLH(c,y[i], y[j], a[i], a[j], lb, rb);
				if (abs(lb - rb) < EPS)
					continue ;
				double ita = 2*kval[i][j] - kval[i][i] -kval[j][j];
				if (ita >= 0)
					continue ;
				a[j] = computeAjAndClipValue(a,y, ei, ej, ita, lb, rb, j);
				if (abs(a[j] - aj_old) < 1e-5)
					continue ;
                a[i] = determineAi(a,y, i,j, aj_old);
                double b_old=b;
                //updateB inlined here for convenience
                {
                    double b1 = b - ei - y[i]*(a[i]-ai_old)*kval[i][i] - y[j]*(a[j]-aj_old)*kval[i][j];
                    double b2 = b - ej - y[i]*(a[i]-ai_old)*kval[i][j] - y[j]*(a[j]-aj_old)*kval[j][j];
                    double finalB = 0;
                    if (0 < a[i] && a[i] < c) finalB = b1;
                    else if (0 < a[j] && a[j] < c) finalB = b2;
                    else finalB = (b1 + b2) / 2;
                    b=finalB;
                }
                num_changed_alphas ++;
                update_eVal(objs,eVal_d,kval_d,b_old,b,i,j,y[i],y[j],ai_old,a[i],aj_old,a[j],eVal);
			}
        }
        // printf("changed: %d\n",num_changed_alphas);
		if (num_changed_alphas == 0) pass ++;
        else pass = 0;
        double ed_clk=(double)clock()/CLOCKS_PER_SEC;
        printf("%d,%d,%f\n",iter,num_changed_alphas,ed_clk-st_clk);
        ++iter;
    }
    *b_out=b;
    free(kval[0]);
    free(kval);
    hipFree(kval_d);
    hipFree(a_d);
    hipFree(y_d);
    
    free(eVal);
    hipFree(eVal_d);
    printf("Training.total time=%lf s\n",(double)clock()/CLOCKS_PER_SEC-init_clk);
}